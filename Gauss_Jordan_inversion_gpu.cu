#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <fstream>
#include <vector>
#include <string>
#include <algorithm>
#pragma comment(lib, "cuda.lib")
#pragma comment(lib, "cudart.lib")
#include <hip/hip_runtime.h>
#include <math.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include ""
#include <hipblas.h>

#define INT_DECIMAL_STRING_SIZE(int_type) ((CHAR_BIT*sizeof(int_type)-1)*10/33+3)


using namespace std;

#define blocksize 32

void display_vector(vector<int> v)
{
	for (int i = 0; i < (int)v.size(); i++)
		cout<< v.at(i) <<" ";
}
void display_vector(vector<float> v)
{
	for (int i = 0; i < (int)v.size(); i++)
		cout<< v.at(i) <<" ";
}

/*storing matrix*/
void matrix_read(string filename,float *L, int dimension){

	FILE *fp;
	int row, col;

	fp = fopen(filename.c_str(), "r");//open output file
	if (fp == NULL)//open failed
		return;

	for (row = 0; row < dimension; row++){
		for (col = 0; col < dimension; col++)
		if (fscanf(fp, "%f,", &L[row * dimension + col]) == EOF) break;//read data

		if (feof(fp)) break;//if the file is over
	}

	fclose(fp);//close file

}

__global__ void nodiag_normalize(float *A, float *I, int n, int i){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < n)
	if (x == i && x!=y){
		I[x*n + y] /= A[i*n + i];
		A[x*n + y] /= A[i*n + i];
	}
	
}

__global__ void diag_normalize(float *A, float *I, int n, int i){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < n)
	if (x == y && x == i){
		I[x*n + y] /= A[i*n + i];
		A[x*n + y] /= A[i*n + i];
	}

}

__global__ void gaussjordan(float *A, float *I, int n, int i)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < n){
		if (x != i){
			I[x*n + y] -= I[i*n + y] * A[x*n + i];
			if (y != i){
				A[x*n + y] -= A[i*n + y] * A[x*n + i];
			}	 
		}
	}

}

__global__ void set_zero(float *A, float *I, int n, int i){
	int x = blockIdx.x * blockDim.x + threadIdx.x;
	int y = blockIdx.y * blockDim.y + threadIdx.y;

	if (x < n && y < n){
		if (x != i){
			if (y == i){
				A[x*n + y] = 0;
			}
		}
	}
}

void savetofile(float *A, string s, int n, int h)
{
	std::ofstream plik;
	plik.open(s);

	for (int j = 0; j<h; j++){
		for (int i = 0; i<h; i++){
			plik << A[j*n + i] << "\t";
		}
		plik << endl;
	}
	plik.close();
}
float execute(string filename,int size){
	const int n = size;
	// creating input
	float *iL = new float[n*n];
	float *L = new float[n*n];

	matrix_read(filename,L, n);
	//savetofile(L, "L.txt", n, n);

	float *d_A, *I, *dI;
	float time;
	hipError_t err;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	int ddsize = n*n*sizeof(float);

	dim3 threadsPerBlock(blocksize, blocksize);
	dim3 numBlocks((n + blocksize - 1) / blocksize, (n + blocksize - 1) / blocksize);
	// memory allocation    
	err = hipMalloc((void**)&d_A, ddsize);
	if (err != hipSuccess){ cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl; }
	err = hipMalloc((void**)&dI, ddsize);
	if (err != hipSuccess){ cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl; }
	I = new float[n*n];

	for (int i = 0; i<n; i++){
		for (int j = 0; j<n; j++){
			if (i == j) I[i*n + i] = 1.0;
			else I[i*n + j] = 0.0;
		}
	}

	//copy data from CPU to GPU
	err = hipMemcpy(d_A, L, ddsize, hipMemcpyHostToDevice);
	if (err != hipSuccess){ cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl; }
	err = hipMemcpy(dI, I, ddsize, hipMemcpyHostToDevice);
	if (err != hipSuccess){ cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl; }

	//timer start
	hipEventRecord(start, 0);

	// L^(-1)    
	for (int i = 0; i<n; i++){
		nodiag_normalize << <numBlocks, threadsPerBlock >> >(d_A, dI, n, i);
		diag_normalize << <numBlocks, threadsPerBlock >> >(d_A, dI, n, i);
		gaussjordan << <numBlocks, threadsPerBlock >> >(d_A, dI, n, i);
		set_zero << <numBlocks, threadsPerBlock >> >(d_A, dI, n, i);
	}

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);

	//copy data from GPU to CPU
	err = hipMemcpy(iL, dI, ddsize, hipMemcpyDeviceToHost);
	if (err != hipSuccess){ cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl; }
	err = hipMemcpy(I, d_A, ddsize, hipMemcpyDeviceToHost);
	if (err != hipSuccess){ cout << hipGetErrorString(err) << " in " << __FILE__ << " at line " << __LINE__ << endl; }

	savetofile(iL, "inv.txt", n, n);
	//savetofile(I, "I.txt", n, n);
	hipFree(d_A);
	hipFree(dI);

	float *c = new float[n*n];
	for (int i = 0; i<n; i++)  
	for (int j = 0; j<n; j++)  
	{
		c[i*n+j] = 0;  //put the initial value to zero
		for (int x = 0; x<n; x++)  
			c[i*n + j] = c[i*n + j] + L[i*n+x] * iL[x*n + j];  //matrix multiplication
	}
	savetofile(c, "c.txt", n, n);

	delete[]I;
	delete[]L;
	delete[]iL;
	return time;
}

char *stringer(int x) {
  int i = x;
  char buf[INT_DECIMAL_STRING_SIZE(int)];
  char *p = &buf[sizeof buf - 1];
  *p = '\0';
  if (i >= 0) {
    i = -i;
  }
  do {
    p--;
    *p = (char) ('0' - i % 10);
    i /= 10;
  } while (i);
  if (x < 0) {
    p--;
    *p = '-';
  }
  size_t len = (size_t) (&buf[sizeof buf] - p);
  char *s = (char*)malloc(len);
  if (s) {
    memcpy(s, p, len);
  }
  return s;
}

int main()
{

	char matrix_types[100][100]={"dense","sparse","hollow","band","identity"};
	execute("input/dense/1000.txt",1000);
	cout<<"\n\nGauss Jordan Inversion GPU Implementation\n\n";
	for(int j=0;j<=4;j++){
		std::vector<float> time;
		std::vector<int> ns;

		for(int i=50;i<=1000;i+=50){
			string num (stringer(i));
			string root ("input/");
			string type (matrix_types[j]);
			string ext (".txt");
			string dir (root+type+"/"+num+ext);
			float t=execute(dir,i);
			ns.push_back(i);
			time.push_back(t);
			cout<<type<<" matrix of size "<<i<<" took "<<t<<" ms\n";
		}
		display_vector(ns);
		cout<<endl;
		display_vector(time);		
		cout<<endl;
	}
}